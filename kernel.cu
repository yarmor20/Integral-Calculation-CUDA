#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <unordered_map>

#include <chrono>
#include <atomic>

#include "inc/config.h"
#include "inc/utils.h"


#define M_PI 3.14159265358979323846 
#define CUDA_CALL(x) { hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA Error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__)); }


__device__ double func_to_integrate(int* m, int* a1, int* a2, int* c, double* x, double* y) {
    double result = 0;
    for (int  i = 0; i < *m; i++) {
        double multiplier = pow(*x - a1[i], 2) + pow(*y - a2[i], 2);
        result += c[i] * exp(-1 / M_PI * multiplier) * cos(M_PI * multiplier);
    }
    return -result;
}


__global__ void integrateKernel(double* intgr_bounds, double* delta, int* m, int* a1, int* a2, int* c, double* res) {
    // get the boundaries for the x and y axis
    double x_start = intgr_bounds[threadIdx.x], x_end = intgr_bounds[threadIdx.x + 1], x = x_start;
    double y_start = intgr_bounds[blockIdx.x], y_end = intgr_bounds[blockIdx.x + 1], y = y_start;

    // integrate
    double temp_res = 0;
    while (x < x_end) {
        while (y < y_end) {
            temp_res += func_to_integrate(m, a1, a2, c, &x, &y);
            y += (*delta);
        }
        y = y_start;
        x += (*delta);
    }
    res[(blockIdx.x) * blockDim.x + (threadIdx.x)] = temp_res * (*delta) * (*delta);
}


int main() {
    std::string config = "config.txt";
    std::unordered_map<std::string, std::string> config_data;

    // Extract program setup
    int res = read_config(config, config_data);
    if (res) { return res; }

    program_setup prg_setup{};
    res = extract_program_args(config_data, prg_setup);
    if (res) { return res; }

    integration_setup intgr_setup{};
    res = extract_integration_setup(config_data, intgr_setup);
    if (res) { return res; }

    // Get block integration bounds
    double* integration_bounds = get_integration_bounds(intgr_setup, prg_setup);
    double delta = fabs(intgr_setup.x[1] - intgr_setup.x[0]) / sqrt(prg_setup.steps);

    // Allocate space for device copies of
    double* d_delta;
    CUDA_CALL(hipMalloc(&d_delta, sizeof(double)));
    CUDA_CALL(hipMemcpy(d_delta, &delta, sizeof(double), hipMemcpyHostToDevice));

    double* d_intgr_bounds;
    CUDA_CALL(hipMalloc(&d_intgr_bounds, (prg_setup.threads + 1) * sizeof(double)));
    CUDA_CALL(hipMemcpy(d_intgr_bounds, integration_bounds, (prg_setup.threads + 1) * sizeof(double), hipMemcpyHostToDevice));

    int* d_m;
    CUDA_CALL(hipMalloc(&d_m, sizeof(int)));
    CUDA_CALL(hipMemcpy(d_m, &intgr_setup.m, sizeof(int), hipMemcpyHostToDevice));

    int* d_a1;
    CUDA_CALL(hipMalloc(&d_a1, intgr_setup.m * sizeof(int)));
    CUDA_CALL(hipMemcpy(d_a1, intgr_setup.a1, intgr_setup.m * sizeof(int), hipMemcpyHostToDevice));

    int* d_a2;
    CUDA_CALL(hipMalloc(&d_a2, intgr_setup.m * sizeof(int)));
    CUDA_CALL(hipMemcpy(d_a2, intgr_setup.a2, intgr_setup.m * sizeof(int), hipMemcpyHostToDevice));

    int* d_c;
    CUDA_CALL(hipMalloc(&d_c, intgr_setup.m * sizeof(int)));
    CUDA_CALL(hipMemcpy(d_c, intgr_setup.c, intgr_setup.m * sizeof(int), hipMemcpyHostToDevice));

    double* d_res;
    CUDA_CALL(hipMalloc(&d_res, prg_setup.threads * prg_setup.threads * sizeof(double)));

    // Call kernel
    integrateKernel <<<prg_setup.threads, prg_setup.threads>>>(d_intgr_bounds, d_delta, d_m, d_a1, d_a2, d_c, d_res);

    // Copy the result back to the host
    double* results = new double[prg_setup.threads * prg_setup.threads];
    hipMemcpy(results, d_res, prg_setup.threads * prg_setup.threads * sizeof(double), hipMemcpyDeviceToHost);

    // Iterate over every partial result (Bad approach) TODO: Rewrite this chunk of code regarding better aproaches
    double result = 0;
    for (int i = 0; i < (prg_setup.threads * prg_setup.threads); i++) {
        result += results[i];
    }

    std::cout << "Result: " << result << std::endl;

    // Free device memory
    hipFree(d_delta); hipFree(d_intgr_bounds), hipFree(d_m);
    hipFree(d_a1); hipFree(d_a2), hipFree(d_c); hipFree(d_res);

    // Free host memory
    free(results);

    return 0;
}